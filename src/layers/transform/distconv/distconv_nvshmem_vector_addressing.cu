#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

/** Copy between two device buffers, using all threads in a warp. */
#define LBANN_TRANSFORM_DISTCONV_NVSHMEM_VECTOR_ADDRESSING_INSTANTIATE
#include "lbann/utils/distconv.hpp"
#include "lbann/base.hpp"

#if defined(LBANN_HAS_NVSHMEM) && defined(LBANN_HAS_DISTCONV)
#include "nvshmem.h"
#include "nvshmemx.h"
#include "lbann/layers/transform/distconv/distconv_nvshmem_vector_addressing.hpp"

namespace distconv{

  namespace{

 __device__ __forceinline__
float atomic_add(float* __restrict__ address,
                 const float val,
                 const int pe){

  int* address_as_int = (int*)address;
  int assumed; 
  int old = nvshmem_int_g(address_as_int, pe);
  do
  {
    assumed = old;
    old = nvshmem_int_atomic_compare_swap(address_as_int, assumed,
                                          __float_as_int(val +
                                                         __int_as_float(assumed)),
                                          pe);
  } while (assumed !=old);
  return __int_as_float(old);
}

__device__ __forceinline__ 
double atomic_add(double* __restrict__ address,
                  const double val,
                  const int pe){

  long long int* address_as_ll = (long long int*)address;
  long long int assumed; 
  long long int old = nvshmem_longlong_g(address_as_ll, pe);
  do
  {
    assumed = old;
    old = nvshmem_longlong_atomic_compare_swap(address_as_ll, assumed,
                                               __double_as_longlong(val +
                                                                    __longlong_as_double(assumed)),
                                               pe);
  }while(assumed != old);
  return __longlong_as_double(old);
}

__device__ __forceinline__
float floor(const float& x) {return floorf(x);}

__device__ __forceinline__
double floor(const double& x) {return floor(x);}

  /**
   * @brief NVSHMEM Gather kernel
   * 
   * workspace(k, j, i) = values(k, indices(k, j), i)
   * 
   */

  template <typename DataType>
  __global__ void Gather_NVSHMEM_Kernel(
    const DataType* __restrict__ shared_buffer,
    const DataType* __restrict__ indices,
    DataType* __restrict__ output,
    const int mini_batch_size,
    const int num_local_values_rows,
    const int num_local_cols,
    const int num_local_output_rows,
    const int pe_group,
    const int pe_stride){
    
    constexpr int warp_size = 32;
    // Indice
    const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
    const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
    
    const size_t nthreadsy = gridDim.y * blockDim.y;
    const size_t nthreadsx = gridDim.x * blockDim.x;
    
    for (size_t mb_i = gidy; mb_i < mini_batch_size; mb_i += nthreadsy){
      // Figure out which rank to send the vector
      const auto mb_offset = mb_i * num_local_cols * num_local_output_rows;
      const auto values_offest = mb_i * num_local_cols * num_local_values_rows;
      const auto ind_offset = mb_i * num_local_output_rows;

      for(size_t row = gidx; row < num_local_output_rows * warp_size; row += nthreadsx){
        const auto ind = static_cast<El::Int>(floor(indices[ind_offset + row / warp_size]));
        if (ind >= 0 ){ 
          const int pe = (pe_group * pe_stride) + (ind / num_local_values_rows);
          const int local_ind = ind % num_local_values_rows;

          nvshmemx_getmem_nbi_warp(&output[mb_offset + (row / warp_size) * num_local_cols],
                                  &shared_buffer[values_offest + local_ind * num_local_cols],
                                  num_local_cols * sizeof(DataType),
                                  pe);
        }
      }
    }
  }

  /**
   * @brief NVSHMEMScatter kernel
   * 
   */
  template <typename DataType>
  __global__ void Scatter_NVSHMEM_Kernel(
      const DataType* __restrict__ values,
      const DataType* __restrict__ indices,
      DataType* __restrict__ outputs,
      const int mini_batch_size,
      const int num_local_values_rows,
      const int num_cols,
      const int num_local_output_rows,
      const int pe_group,
      const int pe_stride){
    // Indices
    const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
    const size_t gidz = threadIdx.z + blockIdx.z * blockDim.z;
    const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;

    const size_t nthreadsx = gridDim.x * blockDim.x;
    const size_t nthreadsy = gridDim.y * blockDim.y;
    const size_t nthreadsz = gridDim.z * blockDim.z;


    for (size_t mb_i = gidz; mb_i < mini_batch_size; mb_i += nthreadsz){
      const auto values_offset = mb_i * num_local_values_rows * num_cols;
      const auto output_offset = mb_i * num_local_output_rows * num_cols;
      const auto indices_offset = mb_i * num_local_values_rows;

      for(size_t row = gidy; row < num_local_values_rows; row += nthreadsy){
        // Figure out which rank to send the vector
        const auto ind = static_cast<El::Int>(floor(indices[indices_offset + row]));
        if (ind >= 0){
          const int pe = (pe_group * pe_stride) + (ind / num_local_output_rows);
          const int local_ind = ind % num_local_output_rows;

          for(size_t i = gidx; i < num_cols; i += nthreadsx){
            const auto val = values[values_offset + row * num_cols + i];
            atomic_add(&outputs[output_offset + (local_ind  * num_cols) + i], val, pe);
          }
        }
      }
    }
  }
  } // namespace <anon>

  namespace tensor{
    
    template<typename DataType>
    void
    ScatterNVSHMEM<DataType>
    ::scatter(const DataType* values,
              const DataType* indices,
              DataType* output,
              const size_t local_mini_batch_size,
              const size_t values_rows_size,
              const size_t values_cols_size,
              const size_t output_rows_size){
                
      const auto buffer_size = local_mini_batch_size * output_rows_size * values_cols_size;
      const auto output_size = local_mini_batch_size * output_rows_size * values_cols_size;
      
      if (buffer_size == 0){ // No work to be done here
        nvshmemx_barrier_all_on_stream(m_stream);
        return ;
      }

      // Make sure the buffers are large enough
      ensure_buffer(buffer_size);
      DataType* output_buffer_ptr = static_cast<DataType*>(m_output_buffer.get()); 
      hipMemsetAsync(output, 0, sizeof(DataType) * output_size, m_stream); // Not sure why this seems to be neccessary
      hipStreamSynchronize(m_stream);
      // To Do: This would result in overcommitted allocation for 
      //        single mini-batch case. Add different thread configuration to 
      //        optimize for the case of single mini_batch - S.Z
      constexpr size_t block_size_x = 1; 
      constexpr size_t block_size_y = 16;  // full-warp
      dim3 block_dims, grid_dims;
      block_dims.x = block_size_x;
      block_dims.y = block_size_y;
      block_dims.z = 1;

      grid_dims.z = (local_mini_batch_size + block_dims.z -1) / block_dims.z;
      grid_dims.y = (values_rows_size + block_dims.y - 1) / block_dims.y;
      grid_dims.x = (values_cols_size + block_dims.x - 1) / block_dims.x;
      sync();
      hipDeviceSynchronize();
      
      Scatter_NVSHMEM_Kernel<<<grid_dims, block_dims, 0, m_stream>>>(values,
                                                                     indices,
                                                                     output_buffer_ptr,
                                                                     local_mini_batch_size,
                                                                     values_rows_size,
                                                                     values_cols_size,
                                                                     output_rows_size,
                                                                     m_group,
                                                                     m_stride);
      hipDeviceSynchronize();
      nvshmemx_quiet_on_stream(m_stream);
      sync();
      // Copy the local workspace buffer onto the output matrix 
      grid_dims.y = (output_rows_size + block_dims.y - 1) / block_dims.y;
      hipMemcpyAsync(output,
                      output_buffer_ptr,
                      sizeof(DataType) * buffer_size,
                      hipMemcpyDeviceToDevice,
                      m_stream);
      hipStreamSynchronize(m_stream);
    }

    template<typename DataType>
    void
    GatherNVSHMEM<DataType>
    ::gather(const DataType* values,
             const DataType* indices,
             DataType* output,
             const size_t local_mini_batch_size,
             const size_t values_rows_size,
             const size_t values_cols_size,
             const size_t output_rows_size){

      const auto buffer_size = local_mini_batch_size * values_rows_size * values_cols_size;
      const auto output_size = local_mini_batch_size * output_rows_size * values_cols_size;
      // The NVSHMEM workspace buffer is the size of the local output matrix 
      if (buffer_size == 0){ // No work to be done here
        nvshmemx_barrier_all_on_stream(m_stream);
        return ;
      }
      ensure_buffer(buffer_size);

      hipMemset(output, 0, sizeof(DataType) * output_size); // Not sure why this seems to be neccessary

      constexpr size_t block_size_x = 32;
      constexpr size_t block_size_y = 16;

      // Copy the local values matrix onto the workspace buffer 
      // The values matrix must be in symmetric heap in order to be remotely accessible 
      // Change grid dimensions for copy kernel. Will be removed in the furure - S.Z

      dim3 block_dims, grid_dims;
      block_dims.x = block_size_x;
      block_dims.y = block_size_y;
      block_dims.z = 1;

      grid_dims.x = (values_cols_size + block_dims.x - 1) / block_dims.x;
      grid_dims.y = (values_rows_size + block_dims.y - 1) / block_dims.y;
      grid_dims.z = (local_mini_batch_size + block_dims.z - 1) / block_dims.z;
          
      hipMemcpyAsync(static_cast<DataType*>(m_output_buffer.get()),
                      values,
                      sizeof(DataType) * buffer_size,
                      hipMemcpyDeviceToDevice,
                      m_stream);
      hipStreamSynchronize(m_stream);
      sync();  // Barrier to ensure all PEs are available

      // To Do: This would result in overcommitted allocation for 
      //        single mini-batch case. Add different thread configuration to 
      //        optimize for the case of single mini_batch - S.Z

      block_dims.x = block_size_x;
      block_dims.y = 1;
      block_dims.z = 1;

      grid_dims.x = (output_rows_size + block_dims.x - 1) / block_dims.x;
      grid_dims.y = (local_mini_batch_size + block_dims.y - 1)/ block_dims.y;
      grid_dims.z = 1;

      Gather_NVSHMEM_Kernel<<<grid_dims, block_dims, 0, m_stream>>>(static_cast<DataType*>(m_output_buffer.get()),
                                                                    indices,
                                                                    output,
                                                                    local_mini_batch_size,
                                                                    values_rows_size,
                                                                    values_cols_size,
                                                                    output_rows_size,
                                                                    m_group,
                                                                    m_stride);
      nvshmemx_quiet_on_stream(m_stream);
      sync();
    }

  
  #define SCATTER_ETI(T)                            \
    template void ScatterNVSHMEM<T>::scatter(       \
      const T* values,                              \
      const T* indices,                             \
      T * output,                                   \
      const size_t local_mini_batch_size,           \
      const size_t values_rows_size,                \
      const size_t values_cols_size,                \
      const size_t output_rows_size);                    

  SCATTER_ETI(float)  
  SCATTER_ETI(double)
  #undef SCATTER_ETI

  #define GATHER_ETI(T)                            \
    template void GatherNVSHMEM<T>::gather(        \
      const T* values,                             \
      const T* indices,                            \
      T * output,                                  \
      const size_t local_mini_batch_size,          \
      const size_t values_rows_size,               \
      const size_t values_cols_size,               \
      const size_t output_rows_size);                    

  GATHER_ETI(float)  
  GATHER_ETI(double)
  #undef GATHER_ETI 

  } // namespace <distconv::tensor>
} //  namespace <distconv>
#endif 
