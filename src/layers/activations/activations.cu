#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_ACTIVATIONS_LAYER_INSTANTIATE
#include "lbann/layers/activations/activations.hpp"
#include "lbann/utils/cuda.hpp"

namespace lbann {

namespace {

// =========================================================
// Operator objects for entry-wise unary layers
// =========================================================
// Note: Unary operator corresponds to forward prop step
// (\f$ y = f(x) \f$) and binary operator corresponds to
// back prop step
// (\f$ \frac{dL}{dx} = \frac{dL}{dy} f'(x) \f$).

/** Log sigmoid operator. */
template <typename TensorDataType>
struct log_sigmoid_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x) const {
    if (x >= El::TypeTraits<TensorDataType>::Zero()) {
      return -cuda::log1p(cuda::exp(-x));
    } else {
      return x - cuda::log1p(cuda::exp(x));
    }
  }
  inline __device__ TensorDataType operator()(const TensorDataType& x, const TensorDataType& dy) const {
    return dy / (El::TypeTraits<TensorDataType>::One() + cuda::exp(x));
  }
};

/** ReLU operator. */
template <typename TensorDataType>
struct relu_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x) const {
    return cuda::max(x, El::TypeTraits<TensorDataType>::Zero());
  }
  inline __device__ TensorDataType operator()(const TensorDataType& x, const TensorDataType& dy) const {
    return x > El::TypeTraits<TensorDataType>::Zero() ? dy : El::TypeTraits<TensorDataType>::Zero();
  }
};

/** SELU operator. */
template <typename TensorDataType>
struct selu_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x) const {
    return (x > El::TypeTraits<TensorDataType>::Zero() ?
            scale * x :
            scale * alpha * cuda::expm1(x));
  }
  inline __device__ TensorDataType operator()(const TensorDataType& x, const TensorDataType& dy) const {
    return (x > El::TypeTraits<TensorDataType>::Zero() ?
            dy * scale :
            dy * scale * alpha * cuda::exp(x));
  }
private:
  static constexpr TensorDataType alpha = 1.6732632423543772848170429916717;
  static constexpr TensorDataType scale = 1.0507009873554804934193349852946;
};

/** Sigmoid operator. */
template <typename TensorDataType>
struct sigmoid_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x) const {
    constexpr TensorDataType one = 1;
    const auto& y = 1 / (one + cuda::exp(-x));
#ifdef LBANN_ENABLE_SIGMOID_CUTOFF
    constexpr TensorDataType eps = cuda::epsilon<TensorDataType>();
    if (y <= eps) { return eps; }
    else if (y >= one - eps) { return one - eps; }
#endif // LBANN_ENABLE_SIGMOID_CUTOFF
    return y;
  }
  inline __device__ TensorDataType operator()(const TensorDataType& x, const TensorDataType& dy) const {
    constexpr TensorDataType one = 1;
    const auto& y = 1 / (one + cuda::exp(-x));
#ifdef LBANN_ENABLE_SIGMOID_CUTOFF
    constexpr TensorDataType eps = cuda::epsilon<TensorDataType>();
    if (y <= eps || y >= El::TypeTraits<TensorDataType>::One() - eps) { return El::TypeTraits<TensorDataType>::Zero(); }
#endif // LBANN_ENABLE_SIGMOID_CUTOFF
    return dy * y * (El::TypeTraits<TensorDataType>::One() - y);
  }
};

/** Softplus operator. */
template <typename TensorDataType>
struct softplus_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x) const {
    if (x > El::TypeTraits<TensorDataType>::Zero()) {
      return cuda::log1p(cuda::exp(-x)) + x;
    } else {
      return cuda::log1p(cuda::exp(x));
    }
  }
  inline __device__ TensorDataType operator()(const TensorDataType& x, const TensorDataType& dy) const {
    return dy / (El::TypeTraits<TensorDataType>::One() + cuda::exp(-x));
  }
};

/** Softsign operator. */
template <typename TensorDataType>
struct softsign_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x) const {
    return x / (El::TypeTraits<TensorDataType>::One() + cuda::abs(x));
  }
  inline __device__ TensorDataType operator()(const TensorDataType& x, const TensorDataType& dy) const {
    const auto& denom = El::TypeTraits<TensorDataType>::One() + cuda::abs(x);
    return dy / (denom * denom);
  }
};

} // namespace

// Template instantiation
#define INSTANTIATE(layer, op)                                          \
  template <typename TensorDataType, data_layout Layout, El::Device Device> \
  void layer<TensorDataType, Layout, Device>::fp_compute() {            \
    cuda::apply_entrywise_unary_operator<op>(                           \
      this->get_prev_activations(),                                     \
      this->get_activations());                                         \
  }                                                                     \
  template <typename TensorDataType, data_layout Layout, El::Device Device> \
  void layer<TensorDataType, Layout, Device>::bp_compute() {            \
    cuda::apply_entrywise_binary_operator<op>(                          \
      this->get_prev_activations(),                                     \
      this->get_prev_error_signals(),                                   \
      this->get_error_signals());                                       \
  }                                                                     \
  UNARY_ETI_INST_MACRO_DEV(layer, El::Device::GPU)

INSTANTIATE(log_sigmoid_layer, log_sigmoid_op);
INSTANTIATE(relu_layer, relu_op);
INSTANTIATE(selu_layer, selu_op);
INSTANTIATE(sigmoid_layer, sigmoid_op);
INSTANTIATE(softplus_layer, softplus_op);
INSTANTIATE(softsign_layer, softsign_op);

} // namespace lbann
